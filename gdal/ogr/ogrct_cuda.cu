#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <bits/huge_val.h>

namespace {

__global__ void degree2radian_wrap_kernel(const int count, double* x, double* y,
                                      const double dfSourceWrapLong) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < count) {
        if (x[i] != HUGE_VAL && y[i] != HUGE_VAL) {
            if (x[i] < dfSourceWrapLong - 180.0)
                x[i] += 360.0;
            else if (x[i] > dfSourceWrapLong + 180)
                x[i] -= 360.0;
        }
        i += gridDim.x * blockDim.x;
    }
}

__global__ void degree2radian_kernel(const int count, double* x, double* y,
                                      const double dfSourceToRadians) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < count) {
        if (x[i] != HUGE_VAL) {
            x[i] *= dfSourceToRadians;
            y[i] *= dfSourceToRadians;
        }
        i += gridDim.x * blockDim.x;
    }
}

__global__ void radian2degree_kernel(const int count, double* x, double* y,
                                      const double dfTargetFromRadians) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < count) {
        if (x[i] != HUGE_VAL && y[i] != HUGE_VAL) {
            x[i] *= dfTargetFromRadians;
            y[i] *= dfTargetFromRadians;
        }
        i += gridDim.x * blockDim.x;
    }
}

__global__ void radian2degree_wrap_kernel(const int count, double* x, double* y,
                                      const double dfTargetWrapLong) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < count) {
        if (x[i] != HUGE_VAL && y[i] != HUGE_VAL) {
            if (x[i] < dfTargetWrapLong - 180.0)
                x[i] += 360.0;
            else if (x[i] > dfTargetWrapLong + 180)
                x[i] -= 360.0;
        }
        i += gridDim.x * blockDim.x;
    }
}

__global__ void check_with_invert_kernel(const int count, double* x, double* y,
                                            const double* x_ori, const double* y_ori,
                                            const double* x_tar, const double* y_tar,
                                            const double dfThreshold) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < count) {
        if (x[i] != HUGE_VAL && y[i] != HUGE_VAL &&
            (fabs(x_tar[i] - x_ori[i]) > dfThreshold ||
             fabs(y_tar[i] - y_ori[i]) > dfThreshold)) {
            x[i] = HUGE_VAL;
            y[i] = HUGE_VAL;
        }
        i += gridDim.x * blockDim.x;
    }
}

__global__ void error_info_kernel(const int count, double* x, double* y,
                                  int* success) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < count) {
        if (x[i] == HUGE_VAL || y[i] == HUGE_VAL)
            success[i] = 0;
        else
            success[i] = 1;

        i += gridDim.x * blockDim.x;
    }
}

}  // anonymous namespace

namespace cu {

void degree2radian_wrap(const int grid_size, const int block_size, const int count,
                    double* x, double* y, const double dfSourceWrapLong) {
    degree2radian_wrap_kernel<<<grid_size, block_size>>>(count, x, y,
                                                     dfSourceWrapLong);
}

void degree2radian(const int grid_size, const int block_size, const int count,
                    double* x, double* y, const double dfSourceToRadians) {
    degree2radian_kernel<<<grid_size, block_size>>>(count, x, y,
                                                     dfSourceToRadians);
}

void radian2degree(const int grid_size, const int block_size, const int count,
                   double* x, double* y, const double dfTargetFromRadians) {
    radian2degree_kernel<<<grid_size, block_size>>>(count, x, y,
                                                    dfTargetFromRadians);
}

void radian2degree_wrap(const int grid_size, const int block_size,
                        const int count, double* x, double* y,
                        const double dfTargetWrapLong) {
    radian2degree_wrap_kernel<<<grid_size, block_size>>>(count, x, y,
                                                         dfTargetWrapLong);
}

void check_with_invert(const int grid_size, const int block_size,
                       const int count, double* x, double* y,
                       const double* x_ori, const double* y_ori,
                       const double* x_tar, const double* y_tar,
                       const double dfThreshold) {
    check_with_invert_kernel<<<grid_size, block_size>>>(
            count, x, y, x_ori, y_ori, x_tar, y_tar, dfThreshold);
}

void error_info(const int grid_size, const int block_size, const int count,
                double* x, double* y, int* success) {
    error_info_kernel<<<grid_size, block_size>>>(count, x, y, success);
}

}  // namespace cu